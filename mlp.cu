#include "hip/hip_runtime.h"
#define CUDA_FILE

#include <stdint.h>
#include "configurations.h"
#include <iostream>

#include "globals.hpp"

using namespace std;

__device__ inline float act_sig(float d){ // 1 / (1 + exp(-x)) // output 0, 1
  return (1.0 / (1.0f + exp(-d)) - 0.5f) * 2.0f;
}
__device__ inline float act_fs(float d){
  return (d / (1.0f + fabsf(d)));
}
__device__ inline float act_sin(float d){
  return sin(d);
}
__device__ inline float act_sinc(float d){
  return sin(d) / d;
}
__device__ inline float act_gauss(float d){
  return (exp(-1.0f * d * d) - 0.5f) * 2.0f;
  //return exp(-1.0f * d * d);
}
__device__ inline float act_relu(float d){
  if (d < 0.0f) return 0.0f;
  else return d;
}
__device__ inline float act_softplus(float d){
    return log(1.0f + exp(d)) - 1.0f;
}

__device__ int layer_sizes_device[layer_sizes_size];
__device__ int layer_starts_w_device[layer_sizes_size - 1];
__device__ int layer_starts_so_device[layer_sizes_size];
__device__ int so_count_for_model_device;
int so_count_for_model;
int w_count_for_model;

#if BW == 0 && COLOUR_TYPE == 1
__device__ inline void convert(uint8_t *colors_in, uint8_t* colors_out)
{
    uint8_t region, remainder, p, q, t;
    uint8_t h = *colors_in;
    uint8_t s = *(colors_in + 1);
    uint8_t v = *(colors_in + 2);
    uint8_t *r = colors_out;
    uint8_t *g = colors_out + 1;
    uint8_t *b = colors_out + 2;

    if (s == 0)
    {
        *r = v;
        *g = v;
        *b = v;
        return;
    }

    region = h / 43;
    remainder = (h - (region * 43)) * 6;

    p = (v * (255 - s)) >> 8;
    q = (v * (255 - ((s * remainder) >> 8))) >> 8;
    t = (v * (255 - ((s * (255 - remainder)) >> 8))) >> 8;

    switch (region)
    {
        case 0:
            *r = v; *g = t; *b = p;
            break;
        case 1:
            *r = q; *g = v; *b = p;
            break;
        case 2:
            *r = p; *g = v; *b = t;
            break;
        case 3:
            *r = p; *g = q; *b = v;
            break;
        case 4:
            *r = t; *g = p; *b = v;
            break;
        default:
            *r = v; *g = p; *b = q;
            break;
    }

    return;
}
#elif BW == 0
__device__ inline void convert(uint8_t *colors_in, uint8_t* colors_out) {
  colors_out[0] = colors_in[2];
  colors_out[1] = colors_in[1];
  colors_out[2] = colors_in[0];
}
#elif BW == 1
__device__ inline void convert(uint8_t *colors_in, uint8_t* colors_out) {
  colors_out[0] = colors_in[0];
}
#endif

#if BW
const int res_count_for_model = 1;
#else
const int res_count_for_model = 4; // BGRA ??
#endif

__device__ inline int get_so_idx(int layer_no, int node_no){
  return layer_starts_so_device[layer_no] + node_no;
}
__device__ inline int get_w_idx(int i, int j, int k){
  //return layer_starts_w_device[k] + i * layer_sizes_device[k + 1] + j;
#ifdef STRANGE_VERSION
  return layer_starts_w_device[k] + i + layer_sizes_device[k]* j;
#else
  return layer_starts_w_device[k] + i + (layer_sizes_device[k] + 1) * j;
#endif
}

__device__ inline void get_y(const int idx, const float x_, const float y_, uint8_t* y, /*float* s, */float* o, const float* w){
  //s[0] = x_;
  //s[1] = y_;
  o[0] = x_;
  o[1] = y_;
  uint8_t network_out[NETWORK_MAX_OUTPUT_SIZE] = { 0 };

#if USE_BIAS
  o[2] = 1; // extra neuron for bias
#endif

  int w_idx = get_w_idx(0, 0, 0);
  for (int k = 1; k < layer_sizes_size - 1; k++){
    int o_idx_ = get_so_idx(k - 1, 0);
    int o_idx_2 = get_so_idx(k, 0);
    int lmt = layer_sizes_device[k];
#if USE_BIAS
    int lmt2 = layer_sizes_device[k - 1] + 1; // if we use bias neurons, increase i's limit by one to add bias coefficient
#else
    int lmt2 = layer_sizes_device[k - 1];
#endif
    for (int j = 0; j < lmt; j++){
      float sum = 0;

      int o_idx = o_idx_;
      for(int i = 0; i < lmt2; i++){
        sum += w[w_idx++] * o[o_idx++];
      }
      //s[get_so_idx(k, j)] = sum;
      switch(k){
        case 1:
          o[o_idx_2++] = ACT1(sum); // hidden layer's neurons are nonlinear
          break;
        case 2:
          o[o_idx_2++] = ACT2(sum); // hidden layer's neurons are nonlinear
          break;
        case 3:
          o[o_idx_2++] = ACT3(sum); // hidden layer's neurons are nonlinear
          break;
        case 4:
          o[o_idx_2++] = ACT4(sum); // hidden layer's neurons are nonlinear
          break;
        case 5:
          o[o_idx_2++] = ACT5(sum); // hidden layer's neurons are nonlinear
          break;
        default:
          o[o_idx_2++] = ACTDEF(sum); // hidden layer's neurons are nonlinear
          break;
      }
    }

#if USE_BIAS
    o[o_idx_2] = 1; // add an extra neuron for bias
#endif
  }
  {
    const int k = layer_sizes_size - 1;
    int o_idx_ = get_so_idx(k - 1, 0);
    int lmt = layer_sizes_device[k];
#if USE_BIAS
    int lmt2 = layer_sizes_device[k - 1] + 1; // if we use bias neurons, increase i's limit by one to add bias coefficient
#else
    int lmt2 = layer_sizes_device[k - 1];
#endif
    for (int j = 0; j < lmt; j++){
      float sum = 0;

      int o_idx = o_idx_;
      for(int i = 0; i < lmt2; i++){
        sum += w[w_idx++] * o[o_idx++];
      }

      // s[get_so_idx(k, j)] = sum; // not necessary!!

      float res = ACTLAST(sum);
      // o[get_so_idx(k, j)] = res; // not necessary!!
      int res_int = (res + 1.0f) * 128;
      uint8_t res_uint = res_int;
      if (res_int > 255) res_uint = 255;
      if (res_int <   0) res_uint =   0;

      network_out[j] = res_uint;
    }

    convert(network_out, y);
#if !BW
    y[3] = 0xffu;
#endif
  }
}

__device__ const int width = WIDTH;
__device__ const int height = HEIGHT;
__device__ const float scale_down = my_min(WIDTH, HEIGHT) / 2.0 * SAMPLE_AREA_RATIO;

extern "C"
__global__ void mlpCUDA(uint8_t* res, /*float* s, */float* o, const float* w, int n, int start)
{
    int ii = blockDim.x * blockIdx.x + threadIdx.x;
    if (ii < n){
      ii += start;

      int x = ii % width;
      int y = ii / width;
      float x_ = (x - width / 2) / scale_down;
      float y_ = (y - height / 2) / scale_down;

      get_y(ii, x_, y_, res + ii * res_count_for_model, /*s + ii * so_count_for_model,*/ o + ii * so_count_for_model_device, w);
    }
}

uint8_t *res_cuda;
float /**s_cuda, */*o_cuda, *w_cuda;
const int stream_count = 1;
hipStream_t streams[stream_count];
void mlp_run_kernel(uint8_t* res, float* w, int n) {
    int threadsPerBlock = 32 * 4;
    int piece = n / stream_count;
    int blocksPerGrid = (piece + threadsPerBlock - 1) / threadsPerBlock;

    hipMemcpy(w_cuda, w, sizeof(float) * w_count_for_model, hipMemcpyHostToDevice);

    for(int i = 0; i < stream_count; i++){
      mlpCUDA<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(res_cuda, /*s_cuda,*/ o_cuda, w_cuda, piece, piece * i);

      // load the answer back into the host
      hipMemcpyAsync(res + sizeof(uint8_t) * piece * i * res_count_for_model, res_cuda + sizeof(uint8_t) * piece * i * res_count_for_model,
                            sizeof(uint8_t) * piece * res_count_for_model, hipMemcpyDeviceToHost, streams[i]);
    }

    hipDeviceSynchronize();
}

__global__ void cuda_init()
{
  layer_starts_so_device[0] = 0;
  layer_starts_w_device[0] = 0;
  for (int i = 1; i < layer_sizes_size; i++){
    layer_starts_so_device[i] = layer_starts_so_device[i - 1] + layer_sizes_device[i - 1] + 1;
  }
  for (int i = 1; i < layer_sizes_size - 1; i++){
    layer_starts_w_device[i] = layer_starts_w_device[i - 1] + (layer_sizes_device[i - 1] + 1) * layer_sizes_device[i];
  }
  so_count_for_model_device = layer_starts_so_device[layer_sizes_size - 1] + layer_sizes_device[layer_sizes_size - 1];
}

void init_kernel(int n) {
  //cout << hipMemcpy(layer_sizes_device, layer_sizes, sizeof(int) * layer_sizes_size, hipMemcpyHostToDevice) << endl;
  cout << hipMemcpyToSymbol(HIP_SYMBOL(layer_sizes_device), layer_sizes, sizeof(int) * layer_sizes_size, 0, hipMemcpyHostToDevice) << endl;
  cout << hipErrorInvalidValue << endl;
  hipDeviceSynchronize();
  cuda_init << <1, 1 >> >();
  cout << "init?" << endl;
  hipDeviceSynchronize();
  cout << "init done" << endl;

  so_count_for_model = 0;
  w_count_for_model = 0;
  for (int i = 0; i < layer_sizes_size; i++)
    so_count_for_model += layer_sizes[i] + 1;
  for (int i = 1; i < layer_sizes_size; i++)
    w_count_for_model += (layer_sizes[i - 1] + 1) * layer_sizes[i];

  so_count_for_model--;

  // allocate and copy memory into the device
  hipMalloc((void **)& res_cuda, sizeof(uint8_t) * n * res_count_for_model);
  //hipMalloc((void **)& s_cuda, sizeof(float) * n * so_count_for_model);
  hipMalloc((void **)& o_cuda, sizeof(float) * n * so_count_for_model);
  hipMalloc((void **)& w_cuda, sizeof(float) * w_count_for_model);

  for(int i = 0; i < stream_count; i++)
    hipStreamCreate(streams + i);

  cout << "init_kernel done" << endl;
}

void end_kernel() {
    hipFree(res_cuda);
    //hipFree(s_cuda);
    hipFree(o_cuda);
    hipFree(w_cuda);

    for(int i = 0; i < stream_count; i++)
      hipStreamDestroy(streams[i]);
}
